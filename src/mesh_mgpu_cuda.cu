#include "hip/hip_runtime.h"

#include "cuda_kernels.h"
#include "debug.h"
#include "error.h"
#include "mesh_mgpu_cuda.h"
#include "minmax.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

EXTERN float *mesh_mgpu_cur_buffer (int gpu)
{
  return cuda_cur_data (gpu);
}

EXTERN void mesh_mgpu_alloc_device_buffer (int gpu, void **buf, size_t size)
{
  hipError_t ret;

  ret = hipSetDevice (cuda_device (gpu));
  check (ret, "hipSetDevice");

  ret = hipMalloc (buf, size);
  check (ret, "hipMalloc");
}

EXTERN void mesh_mgpu_copy_host_to_device (int gpu, void *dest_buffer,
                                           void *src_addr, size_t bytes,
                                           size_t offset_in_bytes)
{
  hipError_t ret;

  hipSetDevice (cuda_device (gpu));
  ret = hipMemcpyAsync ((char *)dest_buffer + offset_in_bytes, src_addr, bytes,
                         hipMemcpyHostToDevice, hip_stream (gpu));
  check (ret, "hipMemcpyAsync");
  hipStreamSynchronize (hip_stream (gpu));
}

EXTERN void mesh_mgpu_copy_device_to_host (int gpu, void *dest_addr,
                                           void *src_buffer, size_t bytes,
                                           size_t offset_in_bytes)
{
  hipError_t ret;

  hipSetDevice (cuda_device (gpu));
  ret = hipMemcpyAsync (dest_addr, (char *)src_buffer + offset_in_bytes, bytes,
                         hipMemcpyDeviceToHost, hip_stream (gpu));
  check (ret, "hipMemcpyAsync");
  hipStreamSynchronize (hip_stream (gpu));
}

EXTERN void mesh_gpu_copy_device_to_device (int gpu, void *dest_buffer,
                                            void *src_buffer, size_t bytes)
{
  hipError_t ret;

  ezp_cuda_event_record (EVENT_START_TRANSFER0, gpu);
  ret = hipMemcpyAsync (dest_buffer,
                         src_buffer, bytes,
                         hipMemcpyDeviceToDevice, hip_stream (gpu));
  check (ret, "hipMemcpyAsync");
  ezp_cuda_event_record (EVENT_END_TRANSFER0, gpu);
}

static __global__ void cuda_gather_cells (float *in, unsigned *indexes,
                                          float *out, unsigned nb)
{
  int index = gpu_get_col ();

  if (index < nb)
    out[index] = in[indexes[index]];
}

EXTERN void mesh_mgpu_launch_cell_gathering_kernel (
    int kernel, int gpu, const size_t threads, const size_t block,
    float *arg0_curbuf, unsigned *arg1_outindex, float *arg2_outval,
    unsigned arg3_outsize)
{
  unsigned grid = threads / block;

  hipSetDevice (cuda_device (gpu));

  ezp_cuda_event_record (EVENT_START_KERNEL0, gpu);
  cuda_gather_cells<<<grid, block, 0, hip_stream (gpu)>>> (
      arg0_curbuf, arg1_outindex, arg2_outval, arg3_outsize);
  ezp_cuda_event_always_record (EVENT_END_KERNEL0, gpu);
}

EXTERN void mesh_mgpu_wait_gathering_kernel (int gpu_wait, int gpu_signal)
{
  ezp_gpu_wait_event (gpu_wait, gpu_signal, EVENT_END_KERNEL0);
}
